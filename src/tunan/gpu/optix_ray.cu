#include "hip/hip_runtime.h"
#include <optix.h>
#include <tunan/gpu/optix_ray.h>

extern "C" {
__constant__ RayParams params;
}

extern "C" __global__ void __raygen__findIntersection() {
    uint3 launch_index = optixGetLaunchIndex();

    // TODO delete
    uchar3 t;
    t.x = 0;
    t.y = 200;
    t.z = 0;
//    t.w = 0;
    params.image[launch_index.y * params.width + launch_index.x] = t;
}

extern "C" __global__ void __closesthit__scene() {

}

extern "C" __global__ void __anyhit__scene() {

}